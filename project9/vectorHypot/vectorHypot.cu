/**
 * Copyright 1993-2013 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 * Extended for use in CS 374 at Calvin College by Joel C. Adams.
 * Extended by Josh Bussis for CS 374 at Calvin University on 12/6/2019 to compute the length of the hypotenuse of 
 * two orthogonal vectors
 */

/**
 * Vector hypotenuse: C = sqrt(A^2 + B^2)
 *
 * This sample is a very basic sample that implements finding the hypotenuse of two orthogonal vectors.
 */

#include <stdio.h>
#include <omp.h>
// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>

/**
 * CUDA Kernel Device code
 *
 * Computes the hypotenuse of A and B
 * The 3 vectors have the same number of elements numElements.
 */
__global__
void vectorHypot(const float *A, const float *B, float *C, unsigned long numElements)
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    if (i < numElements)
    {
        C[i] = sqrt( (A[i]*A[i]) + (B[i]*B[i]) );
    }
}

void checkErr(hipError_t err, const char* msg) 
{
    if (err != hipSuccess)
    {
        fprintf(stderr, "%s (error code %d: '%s')!\n", msg, err, hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
}

/**
 * Host main routine
 */
int main(int argc, char** argv)
{
    // timing variables
    double startCuda, stopCuda;
    double startSeq, stopSeq;
    double startHostDev, stopHostDev;
    double startComp, stopComp;
    double startDevHost, stopDevHost;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Print the vector length to be used, and compute its size
    unsigned long numElements = 50000;
    if (argc == 2) {
      numElements = strtoul( argv[1] , 0, 10 );
    }
    size_t size = numElements * sizeof(float);
    printf("[Hypotenuse of %lu elements]\n", numElements);

    // Allocate the host input vectors A & B
    float * h_A = (float *)malloc(size);
    float * h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float * h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < numElements; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // 1a. Allocate the device input vectors A & B
    float * d_A = NULL;
    err = hipMalloc((void **)&d_A, size);
    checkErr(err, "Failed to allocate device vector A");
    float * d_B = NULL;
    err = hipMalloc((void **)&d_B, size);
    checkErr(err, "Failed to allocate device vector B");

    // 1.b. Allocate the device output vector C
    float * d_C = NULL;
    err = hipMalloc((void **)&d_C, size);
    checkErr(err, "Failed to allocate device vector C");

    // 2. Copy the host input vectors A and B in host memory 
    //     to the device input vectors in device memory
    startCuda = omp_get_wtime();
    startHostDev = omp_get_wtime();
    //printf("Copy input data from the host memory to the CUDA device\n");
    err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    checkErr(err, "Failed to copy device vector A from host to device");


    err = hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    checkErr(err, "Failed to copy device vector B from host to device");
    stopHostDev = omp_get_wtime();

    startComp = omp_get_wtime();
    // 3. Launch the Vector Add CUDA Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid =(numElements + threadsPerBlock - 1) / threadsPerBlock;
    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    vectorHypot<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, numElements);
    err = hipGetLastError();
    checkErr(err, "Failed to launch vectorHypot kernel");
    stopComp = omp_get_wtime();

    startDevHost = omp_get_wtime();
    // 4. Copy the device result vector in device memory
    //     to the host result vector in host memory.
    //printf("Copy output data from the CUDA device to the host memory\n");
    err = hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    checkErr(err, "Failed to copy vector C from device to host");
    stopDevHost = omp_get_wtime();
    stopCuda = omp_get_wtime();
    // Verify that the result vector is correct
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(sqrt( (h_A[i]*h_A[i]) + (h_B[i]*h_B[i]) ) - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("CUDA test PASSED\n");
    printf("CUDA host-to-dev time: %lf\nCUDA comp time: %lf\nCUDA dev-to-host time: %lf\n", stopHostDev - startHostDev, 
                                                                                            stopComp - startComp,
                                                                                            stopDevHost - startDevHost);
    printf("CUDA total time: %lf\n", stopCuda-startCuda); 

    // Free device global memory
    err = hipFree(d_A);
    checkErr(err, "Failed to free device vector A");

    err = hipFree(d_B);
    checkErr(err, "Failed to free device vector B");

    err = hipFree(d_C);
    checkErr(err, "Failed to free device vector C");

    startSeq = omp_get_wtime();
    // repeat the computation sequentially
    for (int i = 0; i < numElements; ++i)
    {
       h_C[i] = sqrt( (h_A[i]*h_A[i]) + (h_B[i]*h_B[i]) );
    }
    stopSeq = omp_get_wtime();

    // verify again
    for (int i = 0; i < numElements; ++i)
    {
        if (fabs(sqrt( (h_A[i]*h_A[i]) + (h_B[i]*h_B[i]) ) - h_C[i]) > 1e-5)
        {
            fprintf(stderr, "Result verification failed at element %d!\n", i);
            exit(EXIT_FAILURE);
        }
    }
    printf("\nNormal test PASSED\n");
    printf("Normal time: %lf\n", stopSeq-startSeq); 
    
    // Free host memory
    free(h_A);
    free(h_B);
    free(h_C);

    // Reset the device and exit
    err = hipDeviceReset();
    checkErr(err, "Unable to reset device");

    printf("Done\n");
    return 0;
}

